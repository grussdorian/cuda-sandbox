#include <stdio.h>
#include <hip/hip_runtime.h>




void print_array(float *A, int N)
{
    for(int i=0;i<N;i++)
        printf("%.2f ",A[i]);
    printf("\n");
}

// generic function to compute global thread id given dim3 for grid and block
__device__
int getGlobalIdx_3D_3D(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) 
    + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void
compute_kernel1(float *input1, float *input2, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);
    //Write code for i
    int i = getGlobalIdx_3D_3D();
    if (i < numElements)
    {
        //Write code for compute
        output[i] = (input1[i]*input2[i])*4;
    }
}


__global__ void
compute_kernel2(float *input, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);
    int i = getGlobalIdx_3D_3D();
    //Write code for i
    if (i < numElements)
    {
        //Write code for compute
        output[i] = (input[i]*input[i]);
    }
}


__global__ void
compute_kernel3(float *input, float *output, int datasize)
{
    int numElements = datasize / sizeof(float);
    //Write code for i
    int i = getGlobalIdx_3D_3D();
    if (i < numElements)
    {
 	   //Write code for compute
        output[i] = input[i]*i;
    }
}



int main(void)
{
    hipError_t err = hipSuccess;

    int numElements = 16384;
    //int numElements = 10;
    size_t size = numElements * sizeof(float);

    float *h_input1 = (float *)malloc(size);

    float *h_input2 = (float *)malloc(size);

    float *h_output1 = (float *)malloc(size);

    float *h_output2 = (float *)malloc(size);

    float *h_output3 = (float *)malloc(size);

    if (h_input1 == NULL || h_input2 == NULL || h_output1 == NULL || h_output2 == NULL || h_output3 == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }


   
    for (int i = 0; i < numElements; ++i)
    {
        scanf("%f",&h_input1[i]);
        
    }
    for (int i = 0; i < numElements; ++i)
    {
        scanf("%f",&h_input2[i]);
        
    }
    


    float *d_input1 = NULL;
    err = hipMalloc((void **)&d_input1, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_input2 = NULL;
    err = hipMalloc((void **)&d_input2, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output1 = NULL;
    err = hipMalloc((void **)&d_output1, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output2 = NULL;
    err = hipMalloc((void **)&d_output2, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_output3 = NULL;
    err = hipMalloc((void **)&d_output3, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector h_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_input1, h_input1, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input1 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_input2, h_input2, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector h_input2 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

  //Complete Code for launching compute_kernel1
    
    dim3 blocksPerGrid1(4,2,2);
    dim3 threadsPerBlock1(32,32,1); 
    compute_kernel1<<<blocksPerGrid1, threadsPerBlock1>>>(d_input1, d_input2, d_output1, size);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel1 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching compute_kernel2
    
    dim3 blocksPerGrid2(2,8,1);
    dim3 threadsPerBlock2(8,8,16); 
    compute_kernel2<<<blocksPerGrid2, threadsPerBlock2>>>(d_output1, d_output2, size);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel2 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Complete Code for launching compute_kernel3 
    dim3 blocksPerGrid3(16,1,1);
    dim3 threadsPerBlock3(128,8,1); 
    compute_kernel3<<<blocksPerGrid3, threadsPerBlock3>>>(d_output2, d_output3, size);

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch process_kernel3 kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_output1, d_output1, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output1 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output2, d_output2, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output2 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_output3, d_output3, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_output3 from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


   
    print_array(h_output3,numElements);
    //print_array(h_output3,numElements);
    

    err = hipFree(d_input1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_input2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_input2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output1);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output2);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_output3);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_output3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(h_input1);
    free(h_input2);
    free(h_output1);
    free(h_output2);
    free(h_output3);

    err = hipDeviceReset();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   
    return 0;
}

