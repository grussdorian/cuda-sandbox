#include <stdio.h>
#include <hip/hip_runtime.h>

#include <math.h>

__global__ void convolution(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_datasize, int mat_dim);
__global__ void convolution2(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_datasize, int mat_dim);

__device__
int getGlobalIdx_3D_3D(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * (blockDim.x * blockDim.y)) 
    + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

// VERTICAL MASK
__global__ void
convolution2(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_datasize, int mat_dim)
{

    mat_conv_input1[4] =0;

    int globalThreadId = getGlobalIdx_3D_3D();

    int row_val = globalThreadId/mat_dim;
    int col_val = globalThreadId%mat_dim;

    if(row_val<mat_dim && col_val<mat_dim)
    {
    int index = mat_dim*row_val + col_val;
	//Write code for convolution

    // corner points

        //top left corner
        if(globalThreadId == 0 ){
            mat_output1[globalThreadId] += mat_input1[globalThreadId] * mat_conv_input1[4]+  (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }
        //bottom left corner
        else if(globalThreadId == mat_dim*(mat_dim -1 ) ) {
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1]);
        }
        //bottom right corner
        else if(globalThreadId == mat_dim*mat_dim - 1 ) {
            mat_output1[globalThreadId] += (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1]);
        }
        //edge points

        //leftmost column
        else if( col_val == 0){
            mat_output1[globalThreadId] += (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1])
                +  (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }
        //top row
        else if(globalThreadId < mat_dim-1 && globalThreadId!=0 ){
            mat_output1[globalThreadId] += (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }
        //rightmost column
        else if(col_val == mat_dim-1 && globalThreadId != mat_dim-1 ){
            mat_output1[globalThreadId] += (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1])
                +  (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }
        //top right corner
        else if(globalThreadId == mat_dim-1){
            mat_output1[globalThreadId] += (mat_input1[index] * mat_conv_input1[4]) + (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }
        //bottom row
        else if(globalThreadId > mat_dim*(mat_dim-1) && globalThreadId < ((mat_dim*mat_dim)-1) ){
             mat_output1[globalThreadId] +=  (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1]);
        } else{
                mat_output1[index] += (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[mat_dim*(row_val-1) + col_val]*mat_conv_input1[1])
                +  (mat_input1[mat_dim*(row_val+1) + col_val]*mat_conv_input1[7]);
        }

    }

}

// HORIZONTAL MASK 
__global__ void
convolution(float *mat_input1, float *mat_conv_input1, float *mat_output1, int mat_datasize, int mat_dim)
{
    int globalThreadId = getGlobalIdx_3D_3D();

    int row_val = globalThreadId/mat_dim;
    int col_val = globalThreadId%mat_dim;

    if(row_val<mat_dim && col_val<mat_dim)
    {
    int index = mat_dim*row_val + col_val;
	//Write code for convolution

    // corner points

        //top left corner
        if(globalThreadId == 0 ){
            mat_output1[globalThreadId] = mat_input1[globalThreadId] * mat_conv_input1[4] + (mat_input1[index +1 ]*mat_conv_input1[5]);
        }
        //bottom left corner
        else if(globalThreadId == mat_dim*(mat_dim -1 ) ) {
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index + 1 ]*mat_conv_input1[5]);
        }
        //bottom right corner
        else if(globalThreadId == mat_dim*mat_dim - 1 ) {
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4])
                 +(mat_input1[index-1]*mat_conv_input1[3]);
        }
        //edge points

        //leftmost column
        else if( col_val == 0){
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index +1 ]*mat_conv_input1[5]);
        }
        //top row
        else if(globalThreadId < mat_dim-1 && globalThreadId!=0 ){
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index -1 ]*mat_conv_input1[3])
                +  (mat_input1[index +1 ]*mat_conv_input1[5]);
        }
        //rightmost column
        else if(col_val == mat_dim-1 && globalThreadId != mat_dim-1 ){
            mat_output1[globalThreadId] =(mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index -1 ]*mat_conv_input1[3]);
        }
        //top right corner
        else if(globalThreadId == mat_dim-1){
            mat_output1[globalThreadId] = (mat_input1[index] * mat_conv_input1[4]) +  (mat_input1[index -1 ]*mat_conv_input1[3]);
        }
        //bottom row
        else if(globalThreadId > mat_dim*(mat_dim-1) && globalThreadId < ((mat_dim*mat_dim)-1) ){
             mat_output1[globalThreadId] =  (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index -1 ]*mat_conv_input1[3])
                +  (mat_input1[index +1 ]*mat_conv_input1[5]);;
        } else{
                mat_output1[index] = (mat_input1[index] * mat_conv_input1[4])
                +  (mat_input1[index -1 ]*mat_conv_input1[3])
                +  (mat_input1[index +1 ]*mat_conv_input1[5]);
        }

    }

}

void print_matrix(float *A,int m,int n)
{
    for(int i =0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            printf("%.2f ",A[i*m+j]);
        printf("\n");
    }
}
int main(void)
{
    hipError_t err = hipSuccess;

    int i, j, k;
    int t;
    scanf("%d",&t);
    while(t--)
    {
        int mat_dim;
        scanf("%d",&mat_dim);
        int mat_num_eles = mat_dim*mat_dim;
        size_t mat_size = mat_num_eles*sizeof(float);


	//populate code for allocating host memory

        float *h_mat_input1 = (float*)malloc(mat_size);
        float *h_mat_output1 = (float*)malloc(mat_size);
        int mat_conv_dim = 3;
        int mat_conv_num_eles = mat_conv_dim*mat_conv_dim;
        size_t mat_conv_size = mat_conv_num_eles*sizeof(float);

        float h_mat_conv_input[] = {0,-1,0,-1,5,-1,0,-1,0};

        if (h_mat_input1 == NULL || h_mat_output1 == NULL || h_mat_conv_input == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        for(i=0;i<mat_num_eles;i++)
        {
            j = i/mat_dim;
            k = i%mat_dim;
            scanf("%f",&h_mat_input1[mat_dim*j + k]);         
	    }

        float *d_mat_input1 = NULL;
        err = hipMalloc((void **)&d_mat_input1, mat_size);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector d_mat_input1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        float *d_mat_conv_input = NULL;
        err = hipMalloc((void **)&d_mat_conv_input, mat_conv_size);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector d_mat_conv_input (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        float *d_mat_output1 = NULL;
        err = hipMalloc((void **)&d_mat_output1, mat_size);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to allocate device vector d_mat_output1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(d_mat_input1, h_mat_input1, mat_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector h_mat_input1 from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipMemcpy(d_mat_conv_input, h_mat_conv_input, mat_conv_size, hipMemcpyHostToDevice);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector h_mat_conv_input from host to device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        // total matrix size upper bounded by 2^26 so size allocated for kernel is adjusted accordingly
        int gx = ceil(mat_dim/32);
        int gy = ceil(mat_dim/32);
        dim3 mat_grid_dim(gx,gy,4);
        // max threads per block = 1024 for backwards compatibility
        dim3 mat_block_dim(32,32,1);
        //successive use of 1D masks
        convolution<<<mat_grid_dim, mat_block_dim>>>(d_mat_input1, d_mat_conv_input, d_mat_output1, mat_size, mat_dim);
        convolution2<<<mat_grid_dim, mat_block_dim>>>(d_mat_input1, d_mat_conv_input, d_mat_output1, mat_size, mat_dim);

       
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch process_kernel2 kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }


        err = hipMemcpy(h_mat_output1, d_mat_output1, mat_size, hipMemcpyDeviceToHost);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector d_mat_output1 from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        
  
        err = hipFree(d_mat_input1);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_input1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipFree(d_mat_conv_input);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_conv_input (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        err = hipFree(d_mat_output1);
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to free device vector d_mat_output1 (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
       print_matrix(h_mat_output1,mat_dim,mat_dim);
        
        free(h_mat_input1);
        free(h_mat_output1);

        err = hipDeviceReset();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    return 0;
}
